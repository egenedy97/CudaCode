#include "hip/hip_runtime.h"
#include <stdio.h>
#include <bits/stdc++.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512
__global__ void add(int *a, int *b, int *c, int n) {
   int index = threadIdx.x+ blockIdx.x* blockDim.x;
   if (index < n)c[index] = a[index] + b[index];
}

void random_ints ( int *a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}

int main(void) 
{
   int*a, *b, *c;
   // host copies of a, b, c
   int*d_a, *d_b, *d_c;
   // device copies of a, b, c
   int size = N * sizeof(int);
   // Allocspace for device copies of a, b, c
   hipMalloc((void**)&d_a, size);
   hipMalloc((void**)&d_b, size);
   hipMalloc((void**)&d_c, size);
   // Allocspace for host copies of a, b, c and setup input values
   a = (int*)malloc(size);
   random_ints(a, N);
   b = (int*)malloc(size); 
   random_ints(b, N);
   c = (int*)malloc(size);
   hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
   hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
   add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
   hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
   free(a); free(b); free(c);
   hipFree(d_a); hipFree(d_b); hipFree(d_c);
   return 0;
}