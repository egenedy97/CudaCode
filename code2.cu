
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <bits/stdc++.h>
__global__ void add(int*a, int*b, int*c) {*c = *a + *b;}

int main(void) {
    int a, b, c;// host copies of a, b, c
    int*d_a, *d_b, *d_c;// device copies of a, b, c
    int size = sizeof(int);// Allocate space for device copies of a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);// Setup input values
    a = 2;
    b = 5;
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    add<<<1,1>>>(d_a, d_b, d_c);
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("%d\n%d\n%d\n" , c, a,b );
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0 ; 
}
